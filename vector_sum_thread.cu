
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    while(tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}


int main(void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    int i, x;

    hipMalloc((void**)&dev_a,N*sizeof(int));
    hipMalloc((void**)&dev_b,N*sizeof(int));
    hipMalloc((void**)&dev_c,N*sizeof(int));
    

    for(i=0; i<N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<(N+127)/128, 128>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost); 

    for(x=0; x<N; x++)
    {
        printf("%d + %d = %d\n", a[x], b[x], c[x]);
    } 
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;   

}